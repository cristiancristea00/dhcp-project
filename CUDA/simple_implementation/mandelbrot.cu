#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

#define MAX_COLOR 255 // Maximum color intensity for grayscale (used for coloring the fractal)


// Complex plane boundaries for Mandelbrot set
#define X_MIN -2.0f   // Minimum real      part (left   boundary of the complex plane)
#define X_MAX  1.0f   // Maximum real      part (right  boundary of the complex plane)
#define Y_MIN -1.2f   // Minimum imaginary part (bottom boundary of the complex plane)
#define Y_MAX  1.2f   // Maximum imaginary part (top    boundary of the complex plane)

auto start_chrono_generateFractal          = std::chrono::high_resolution_clock::now();
auto start_chrono_new                      = std::chrono::high_resolution_clock::now();
auto start_chrono_cudaMalloc               = std::chrono::high_resolution_clock::now();
auto start_chrono_blockSize                = std::chrono::high_resolution_clock::now();
auto start_chrono_gridSize                 = std::chrono::high_resolution_clock::now();
auto start_chrono_generateMandelbrotKernel = std::chrono::high_resolution_clock::now();
auto start_chrono_cudaDeviceSynchronize    = std::chrono::high_resolution_clock::now();
auto start_chrono_cudaEventSynchronize     = std::chrono::high_resolution_clock::now();
auto start_chrono_cudaMemcpy               = std::chrono::high_resolution_clock::now();
auto start_chrono_coloredImage             = std::chrono::high_resolution_clock::now();
auto start_chrono_imwrite                  = std::chrono::high_resolution_clock::now();
auto start_chrono_cudaFree                 = std::chrono::high_resolution_clock::now();
auto start_chrono_delete                   = std::chrono::high_resolution_clock::now();

auto stop_chrono_generateFractal           = std::chrono::high_resolution_clock::now();
auto stop_chrono_new                       = std::chrono::high_resolution_clock::now();
auto stop_chrono_cudaMalloc                = std::chrono::high_resolution_clock::now();
auto stop_chrono_blockSize                 = std::chrono::high_resolution_clock::now();
auto stop_chrono_gridSize                  = std::chrono::high_resolution_clock::now();
auto stop_chrono_generateMandelbrotKernel  = std::chrono::high_resolution_clock::now();
auto stop_chrono_cudaDeviceSynchronize     = std::chrono::high_resolution_clock::now();
auto stop_chrono_cudaEventSynchronize      = std::chrono::high_resolution_clock::now();
auto stop_chrono_cudaMemcpy                = std::chrono::high_resolution_clock::now();
auto stop_chrono_coloredImage              = std::chrono::high_resolution_clock::now();
auto stop_chrono_imwrite                   = std::chrono::high_resolution_clock::now();
auto stop_chrono_cudaFree                  = std::chrono::high_resolution_clock::now();
auto stop_chrono_delete                    = std::chrono::high_resolution_clock::now();


// Struct to hold image size (width and height)
struct ImageSize
{
   int width;  // Image width  in pixels
   int height; // Image height in pixels
};


// CUDA device function: calculates the Mandelbrot value for a given pixel
__device__ uint8_t mandelbrot(uint32_t x, uint32_t y, ImageSize size, float x_min, float x_max, float y_min, float y_max, uint32_t max_iterations)
{
   // Map pixel coordinates (x, y) to complex plane coordinates (real and imaginary parts)
   // The resolution of the image is mapped to the range defined by X_MIN, X_MAX, Y_MIN, Y_MAX
   float real = x_min + (x_max - x_min) * x / size.width;  // x-coordinate mapped to the real      part of the complex number
   float imag = y_min + (y_max - y_min) * y / size.height; // y-coordinate mapped to the imaginary part of the complex number
   
   // Initialize z = 0 + 0i (starting point for the iteration process)
   float zr = 0.0f, zi = 0.0f;
   uint32_t iterations = 0; // Counter for the number of iterations before escaping
   
   // Iterative process for Mandelbrot set: z = z^2 + c
   // If the magnitude of z exceeds 2 (i.e., zr^2 + zi^2 > 4), the point escapes to infinity
   // and is considered outside the Mandelbrot set
   
   while(iterations < max_iterations)
   {
      float zr2 = zr * zr - zi * zi + real; // Real      part of the new z
      float zi2 = 2.0f * zr * zi + imag;    // Imaginary part of the new z
      
      zr = zr2; // Update the real      part of z
      zi = zi2; // Update the imaginary part of z
      
      // Escape condition: if the magnitude of z exceeds 2, the point escapes to infinity
      if(zr * zr + zi * zi > 4.0f)
      {
         break; // Exit the loop if the point escapes (it’s not part of the Mandelbrot set)
      }
      ++iterations; // Increment the iteration counter
   }
   
   // Color mapping:
   // If the point didn't escape (iterations == max_iterations), it’s part of the Mandelbrot set and is colored black (0)
   // If the point escaped, the color depends on how quickly it escaped (this is determined by the iteration count)
   if(iterations == max_iterations)
   {
      return 0; // Points inside the Mandelbrot set are black (color 0)
   }
   
   // Use a logarithmic scale for coloring based on the escape time (iteration count)
   // The longer the point takes to escape, the more intense its color will be
   return static_cast<uint8_t>(MAX_COLOR * logf(static_cast<float>(iterations + 1)) / logf(static_cast<float>(max_iterations)));
}


// CUDA kernel: This function runs on the GPU and calculates Mandelbrot fractal for each pixel
__global__ void generateMandelbrotKernel(uint8_t *image, ImageSize size, float x_min, float x_max, float y_min, float y_max, uint32_t max_iterations)
{
   // Calculate the pixel coordinates (x, y) for this thread in the image
   uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // x-coordinate of the pixel (thread index in x)
   uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // y-coordinate of the pixel (thread index in y)
   
   // Check if the current thread is within the image bounds (to avoid out-of-bounds memory access)
   if(x < size.width && y < size.height)
   {
      // Call the Mandelbrot device function to calculate the color for the pixel at (x, y)
      uint8_t color = mandelbrot(x, y, size, x_min, x_max, y_min, y_max, max_iterations);
      
      // Store the calculated color in the image buffer (image[y * width + x] represents the pixel location)
      image[y * size.width + x] = color;
   }
}


// Function to generate the Mandelbrot fractal on the GPU and save the result as an image
void generateFractal(ImageSize size, uint32_t max_iterations)
{
   // Declare and create CUDA events to measure GPU execution time
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   uint8_t *d_image;                                         // Pointer to image data in device memory (GPU)
   
   start_chrono_new = std::chrono::high_resolution_clock::now();
   uint8_t *h_image = new uint8_t[size.width * size.height]; // Pointer to image data in host   memory (CPU)
   stop_chrono_new = std::chrono::high_resolution_clock::now();
   
   start_chrono_cudaMalloc = std::chrono::high_resolution_clock::now();
   // Step 1: Allocate memory on the GPU
   // The GPU will store the image as a 1D array of uint8_t values. Each value represents a pixel's grayscale intensity
   // The size of the memory allocated is proportional to the image's dimensions: width * height * size of each pixel (1 byte)
   hipMalloc((void**)&d_image, size.width * size.height * sizeof(uint8_t));
   stop_chrono_cudaMalloc = std::chrono::high_resolution_clock::now();
   
   start_chrono_blockSize = std::chrono::high_resolution_clock::now();
   // Step 2: Define the CUDA thread block size
   // A block contains a 2D grid of threads. Here, we use 8x8 threads per block, for a total of 64 pixels processed in parallel per block
   // This block size is chosen to balance GPU memory usage and computational efficiency, though it can be adjusted for optimization
   dim3 blockSize(8, 8); // A block processes an 8x8 section of the image
   stop_chrono_blockSize = std::chrono::high_resolution_clock::now();
   
   start_chrono_gridSize = std::chrono::high_resolution_clock::now();
   // Step 3: Calculate the grid size
   // The grid determines how many blocks are needed to cover the entire image
   // Each dimension of the grid is calculated by dividing the image dimensions by the block size, with rounding up to handle edge cases
   // The "+ blockSize.x - 1" ensures that the grid size is rounded up if the image dimensions aren't divisible by the block size
   dim3 gridSize(
      (size.width  + blockSize.x - 1) / blockSize.x, // Total blocks needed along the image width
      (size.height + blockSize.y - 1) / blockSize.y  // Total blocks needed along the image height
   );
   stop_chrono_gridSize = std::chrono::high_resolution_clock::now();
   
   std::cout << "Total threads per block = 8×8 = 64 threads\n";
   std::cout << "Each thread is responsible for computing one pixel of the image, meaning each block processes a 64 pixel region in parallel.\n\n";
   std::cout << "The grid is a 2D arrangement of blocks, and its size is calculated to cover the entire image.\n";
   std::cout << "The number of blocks required for the width and height of the image is computed as follows:\n";
   std::cout << "Total blocks needed along the image width:  " << (size.width  + blockSize.x - 1) / blockSize.x << std::endl;
   std::cout << "Total blocks needed along the image height: " << (size.height + blockSize.y - 1) / blockSize.y << std::endl << std::endl;
   
   // Record the start time for GPU execution using cudaEvent
   hipEventRecord(start);
   
   start_chrono_generateMandelbrotKernel = std::chrono::high_resolution_clock::now();
   // Step 4: Launch the CUDA kernel
   // The generateMandelbrotKernel function is executed in parallel across all threads in the grid
   // Each thread computes the Mandelbrot set for one pixel in the image
   generateMandelbrotKernel<<<gridSize, blockSize>>>(d_image, size, X_MIN, X_MAX, Y_MIN, Y_MAX, max_iterations);
   stop_chrono_generateMandelbrotKernel = std::chrono::high_resolution_clock::now();
   
   start_chrono_cudaDeviceSynchronize = std::chrono::high_resolution_clock::now();
   // Step 5: Synchronize the device
   // Ensures that all threads finish their computation before moving to the next step
   // Without this, the program may proceed prematurely, potentially causing incorrect or incomplete results
   hipDeviceSynchronize();
   stop_chrono_cudaDeviceSynchronize = std::chrono::high_resolution_clock::now();
   
   // Record the stop time for GPU execution using cudaEvent
   hipEventRecord(stop);
   
   start_chrono_cudaEventSynchronize = std::chrono::high_resolution_clock::now();
   // Synchronize to ensure all GPU operations are complete before measuring time
   hipEventSynchronize(stop);
   stop_chrono_cudaEventSynchronize = std::chrono::high_resolution_clock::now();
   
   float elapsed_cuda = 0;
   hipEventElapsedTime(&elapsed_cuda, start, stop);
   std::cout << "Execution time using cudaEvent:   " << elapsed_cuda << " ms\n\n";
   
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   // Check for any errors that occurred during the kernel execution
   hipError_t err = hipGetLastError();
   if(err != hipSuccess)
   {
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
   }
   
   start_chrono_cudaMemcpy = std::chrono::high_resolution_clock::now();
   // Copy the resulting image from GPU memory back to CPU memory (host)
   hipMemcpy(h_image, d_image, size.width * size.height * sizeof(uint8_t), hipMemcpyDeviceToHost);
   stop_chrono_cudaMemcpy = std::chrono::high_resolution_clock::now();
   
   start_chrono_coloredImage = std::chrono::high_resolution_clock::now();
   // Create an OpenCV matrix (image) from the raw pixel data (grayscale image)
   cv::Mat img(size.height, size.width, CV_8UC1, h_image);
   
   // Apply a colormap (e.g., MAGMA) to enhance the visualization of the fractal
   cv::Mat coloredImage;
   cv::applyColorMap(img, coloredImage, cv::COLORMAP_MAGMA);
   stop_chrono_coloredImage = std::chrono::high_resolution_clock::now();
   
   start_chrono_imwrite = std::chrono::high_resolution_clock::now();
   // Save the generated fractal image as a PNG file
   cv::imwrite("Mandelbrot_CUDA.png", coloredImage);
   stop_chrono_imwrite = std::chrono::high_resolution_clock::now();
   
   start_chrono_cudaFree = std::chrono::high_resolution_clock::now();
   // Free the allocated memory on both GPU and CPU
   hipFree(d_image);
   stop_chrono_cudaFree = std::chrono::high_resolution_clock::now();
   
   start_chrono_delete = std::chrono::high_resolution_clock::now();
   delete[] h_image;
   stop_chrono_delete = std::chrono::high_resolution_clock::now();
}


//*****************************************************************************
// a) cudaEvent
//    Measures the actual execution time of the kernel on the GPU.
//    It excludes data transfer times between the CPU (host) and GPU (device)
//    as well as other initialization operations.
//    It is precise because it directly synchronizes with the GPU.
//
// b) std::chrono
//    Measures the total execution time, including:
//    - Kernel launch (which is typically asynchronous).
//    - Data transfer times between the CPU and GPU.
//    - Any other operations running on the CPU, such as CUDA API calls.
//    - Possible delays caused by synchronization between the CPU and GPU
//    (e.g., hipDeviceSynchronize() waits for all GPU operations to complete).
//*****************************************************************************


int main(int argc, char *argv[])
{
   // Check if the correct number of arguments is passed to the program
   if(argc != 4)
   {
      std::cerr << "Usage: " << argv[0] << " <width> <height> <max_iterations> " << std::endl;
      return EXIT_FAILURE;
   }
   
   // Parse command-line arguments for image width, height, and maximum iterations
   int width  = std::stoi(argv[1]); // Image width  in pixels
   int height = std::stoi(argv[2]); // Image height in pixels
   uint32_t max_iterations = std::stoi(argv[3]); // Maximum iterations for Mandelbrot calculation
   
   // Store image size in a structure
   ImageSize size = {width, height};
   
   start_chrono_generateFractal = std::chrono::high_resolution_clock::now();
   // Call the function to generate the Mandelbrot fractal on the GPU and save the result as an image
   generateFractal(size, max_iterations);
   stop_chrono_generateFractal = std::chrono::high_resolution_clock::now();
   
   std::chrono::duration<double, std::milli> elapsed_chrono_generateFractal              = stop_chrono_generateFractal          - start_chrono_generateFractal;
   std::chrono::duration<double, std::milli> elapsed_chrono_new                          = stop_chrono_new                      - start_chrono_new;
   std::chrono::duration<double, std::milli> elapsed_chrono_cudaMalloc                   = stop_chrono_cudaMalloc               - start_chrono_cudaMalloc;
   std::chrono::duration<double, std::milli> elapsed_chrono_blockSize                    = stop_chrono_blockSize                - start_chrono_blockSize;
   std::chrono::duration<double, std::milli> elapsed_chrono_gridSize                     = stop_chrono_gridSize                 - start_chrono_gridSize;
   std::chrono::duration<double, std::milli> elapsed_chrono_generateMandelbrotKernel     = stop_chrono_generateMandelbrotKernel - start_chrono_generateMandelbrotKernel;
   std::chrono::duration<double, std::milli> elapsed_chrono_cudaDeviceSynchronize        = stop_chrono_cudaDeviceSynchronize    - start_chrono_cudaDeviceSynchronize;
   std::chrono::duration<double, std::milli> elapsed_chrono_generateMandelbrotKernelSync = stop_chrono_cudaDeviceSynchronize    - start_chrono_generateMandelbrotKernel;
   std::chrono::duration<double, std::milli> elapsed_chrono_cudaEventSynchronize         = stop_chrono_cudaEventSynchronize     - start_chrono_cudaEventSynchronize;
   std::chrono::duration<double, std::milli> elapsed_chrono_cudaMemcpy                   = stop_chrono_cudaMemcpy               - start_chrono_cudaMemcpy;
   std::chrono::duration<double, std::milli> elapsed_chrono_coloredImage                 = stop_chrono_coloredImage             - start_chrono_coloredImage;
   std::chrono::duration<double, std::milli> elapsed_chrono_imwrite                      = stop_chrono_imwrite                  - start_chrono_imwrite;
   std::chrono::duration<double, std::milli> elapsed_chrono_cudaFree                     = stop_chrono_cudaFree                 - start_chrono_cudaFree;
   std::chrono::duration<double, std::milli> elapsed_chrono_delete                       = stop_chrono_delete                   - start_chrono_delete;
   std::chrono::duration<double, std::milli> elapsed_chrono_delete_new                   = stop_chrono_delete                   - start_chrono_new;
   
   std::cout << "Execution time using std::chrono: for generateFractal                                 " << std::setw(10) << elapsed_chrono_generateFractal.count()              << " ms <==" << std::endl;
   std::cout << "Execution time using std::chrono: for new                                             " << std::setw(10) << elapsed_chrono_new.count()                          << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for hipMalloc                                      " << std::setw(10) << elapsed_chrono_cudaMalloc.count()                   << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for blockSize                                       " << std::setw(10) << elapsed_chrono_blockSize.count()                    << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for gridSize                                        " << std::setw(10) << elapsed_chrono_gridSize.count()                     << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for generateMandelbrotKernel                        " << std::setw(10) << elapsed_chrono_generateMandelbrotKernel.count()     << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for hipDeviceSynchronize                           " << std::setw(10) << elapsed_chrono_cudaDeviceSynchronize.count()        << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for generateMandelbrotKernel + hipDeviceSynchronize" << std::setw(10) << elapsed_chrono_generateMandelbrotKernelSync.count() << " ms !!! "<< std::endl;
   std::cout << "Execution time using std::chrono: for hipEventSynchronize                            " << std::setw(10) << elapsed_chrono_cudaEventSynchronize.count()         << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for hipMemcpy                                      " << std::setw(10) << elapsed_chrono_cudaMemcpy.count()                   << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for coloredImage                                    " << std::setw(10) << elapsed_chrono_coloredImage.count()                 << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for imwrite                                         " << std::setw(10) << elapsed_chrono_imwrite.count()                      << " ms <--" << std::endl;
   std::cout << "Execution time using std::chrono: for hipFree                                        " << std::setw(10) << elapsed_chrono_cudaFree.count()                     << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for delete                                          " << std::setw(10) << elapsed_chrono_delete.count()                       << " ms"     << std::endl;
   std::cout << "Execution time using std::chrono: for delete - new (actual time in generateFractal)   " << std::setw(10) << elapsed_chrono_delete_new.count()                   << " ms"     << std::endl;
   
   return EXIT_SUCCESS;
}
