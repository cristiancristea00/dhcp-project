#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <cmath>

#define MAX_COLOR 255 // Maximum color intensity for grayscale (used for coloring the fractal)


// Complex plane boundaries for Cosine set
#define X_MIN -2.0f   // Minimum real      part (left   boundary of the complex plane)
#define X_MAX  5.0f   // Maximum real      part (right  boundary of the complex plane)
#define Y_MIN -2.0f   // Minimum imaginary part (bottom boundary of the complex plane)
#define Y_MAX  2.0f   // Maximum imaginary part (top    boundary of the complex plane)
#define RADIUS 31.4f  // float radius = 10.0f * M_PI;


// Struct to hold image size (width and height)
struct ImageSize
{
   int width;  // Image width  in pixels
   int height; // Image height in pixels
};


// CUDA device function: calculates the Cosine value for a given pixel
__device__ uint8_t cosine(uint32_t x, uint32_t y, ImageSize size, float x_min, float x_max, float y_min, float y_max, float radius, uint32_t max_iterations)
{
   // Map pixel coordinates (x, y) to complex plane coordinates (real and imaginary parts)
   // The resolution of the image is mapped to the range defined by X_MIN, X_MAX, Y_MIN, Y_MAX
   float real = x_min + (x_max - x_min) * x / size.width;  // x-coordinate mapped to the real      part of the complex number
   float imag = y_min + (y_max - y_min) * y / size.height; // y-coordinate mapped to the imaginary part of the complex number
   
   // Initialize z = 0 + 0i (starting point for the iteration process)
   float zr = 0.0f, zi = 0.0f;
   uint32_t iterations = 0; // Counter for the number of iterations before escaping
   
    while(iterations < max_iterations)
   {
      float zr2 =  cosf(zr) * coshf(zi) + real; // Real      part of the new z
      float zi2 = -sinf(zr) * sinhf(zi) + imag; // Imaginary part of the new z
      
      zr = zr2; // Update the real      part of z
      zi = zi2; // Update the imaginary part of z
      
      // Escape condition: if the magnitude of z exceeds radius
      if(zr * zr + zi * zi > radius * radius)
      {
         break; // Exit the loop if the point escapes (it’s not part of the Cosine set)
      }
      ++iterations; // Increment the iteration counter
   }
   
   // Color mapping:
   // If the point didn't escape (iterations == max_iterations), it’s part of the Cosine set and is colored black (0)
   // If the point escaped, the color depends on how quickly it escaped (this is determined by the iteration count)
   if(iterations == max_iterations)
   {
      return 0; // Points inside the Cosine set are black (color 0)
   }
   
   // Use a logarithmic scale for coloring based on the escape time (iteration count)
   // The longer the point takes to escape, the more intense its color will be
   return static_cast<uint8_t>(MAX_COLOR * logf(static_cast<float>(iterations + 1)) / logf(static_cast<float>(max_iterations)));
}


// CUDA kernel: This function runs on the GPU and calculates Cosine fractal for each pixel
__global__ void generateCosineKernel(uint8_t *image, ImageSize size, float x_min, float x_max, float y_min, float y_max, float radius, uint32_t max_iterations)
{
   // Calculate the pixel coordinates (x, y) for this thread in the image
   uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // x-coordinate of the pixel (thread index in x)
   uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // y-coordinate of the pixel (thread index in y)
   
   // Check if the current thread is within the image bounds (to avoid out-of-bounds memory access)
   if(x < size.width && y < size.height)
   {
      // Call the Cosine device function to calculate the color for the pixel at (x, y)
      uint8_t color = cosine(x, y, size, x_min, x_max, y_min, y_max, radius, max_iterations);
      
      // Store the calculated color in the image buffer (image[y * width + x] represents the pixel location)
      image[y * size.width + x] = color;
   }
}


// Function to generate the Cosine fractal on the GPU and save the result as an image
void generateFractal(ImageSize size, uint32_t max_iterations)
{
   // Declare and create CUDA events to measure GPU execution time
   hipEvent_t start, stop;
   hipEventCreate(&start);
   hipEventCreate(&stop);
   
   uint8_t *d_image;                                         // Pointer to image data in device memory (GPU)
   uint8_t *h_image = new uint8_t[size.width * size.height]; // Pointer to image data in host   memory (CPU)
   
   // Step 1: Allocate memory on the GPU
   // The GPU will store the image as a 1D array of uint8_t values. Each value represents a pixel's grayscale intensity
   // The size of the memory allocated is proportional to the image's dimensions: width * height * size of each pixel (1 byte)
   hipMalloc((void**)&d_image, size.width * size.height * sizeof(uint8_t));
   
   // Step 2: Define the CUDA thread block size
   // A block contains a 2D grid of threads. Here, we use 8x8 threads per block, for a total of 64 pixels processed in parallel per block
   // This block size is chosen to balance GPU memory usage and computational efficiency, though it can be adjusted for optimization
   dim3 blockSize(16, 16); // A block processes an 8x8 section of the image
   
   // Step 3: Calculate the grid size
   // The grid determines how many blocks are needed to cover the entire image
   // Each dimension of the grid is calculated by dividing the image dimensions by the block size, with rounding up to handle edge cases
   // The "+ blockSize.x - 1" ensures that the grid size is rounded up if the image dimensions aren't divisible by the block size
   dim3 gridSize(
      (size.width  + blockSize.x - 1) / blockSize.x, // Total blocks needed along the image width
      (size.height + blockSize.y - 1) / blockSize.y  // Total blocks needed along the image height
   );
   
   std::cout << "Total threads per block = 16x16 = 256 threads\n";
   std::cout << "Each thread is responsible for computing one pixel of the image, meaning each block processes a 64 pixel region in parallel.\n\n";
   std::cout << "The grid is a 2D arrangement of blocks, and its size is calculated to cover the entire image.\n";
   std::cout << "The number of blocks required for the width and height of the image is computed as follows:\n";
   std::cout << "Total blocks needed along the image width:  " << (size.width  + blockSize.x - 1) / blockSize.x << std::endl;
   std::cout << "Total blocks needed along the image height: " << (size.height + blockSize.y - 1) / blockSize.y << std::endl << std::endl;
   
   // Record the start time for GPU execution using cudaEvent
   hipEventRecord(start);
   
   // Step 4: Launch the CUDA kernel
   // The generateCosineKernel function is executed in parallel across all threads in the grid
   // Each thread computes the Cosine set for one pixel in the image
   generateCosineKernel<<<gridSize, blockSize>>>(d_image, size, X_MIN, X_MAX, Y_MIN, Y_MAX, RADIUS, max_iterations);
   
   // Step 5: Synchronize the device
   // Ensures that all threads finish their computation before moving to the next step
   // Without this, the program may proceed prematurely, potentially causing incorrect or incomplete results
   hipDeviceSynchronize();
   
   // Record the stop time for GPU execution using cudaEvent
   hipEventRecord(stop);
   // Synchronize to ensure all GPU operations are complete before measuring time
   hipEventSynchronize(stop);
   
   float elapsed_cuda = 0;
   hipEventElapsedTime(&elapsed_cuda, start, stop);
   std::cout << "Execution time using cudaEvent:   " << elapsed_cuda << " ms" << std::endl;
   
   hipEventDestroy(start);
   hipEventDestroy(stop);
   
   // Check for any errors that occurred during the kernel execution
   hipError_t err = hipGetLastError();
   if(err != hipSuccess)
   {
      std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
   }
   
   // Copy the resulting image from GPU memory back to CPU memory (host)
   hipMemcpy(h_image, d_image, size.width * size.height * sizeof(uint8_t), hipMemcpyDeviceToHost);
   
   // Create an OpenCV matrix (image) from the raw pixel data (grayscale image)
   cv::Mat img(size.height, size.width, CV_8UC1, h_image);
   
   // Apply a colormap (e.g., MAGMA) to enhance the visualization of the fractal
   cv::Mat coloredImage;
   cv::applyColorMap(img, coloredImage, cv::COLORMAP_MAGMA);
   
   // Save the generated fractal image as a PNG file
   cv::imwrite("img_fractals/CUDA_cosine.png", coloredImage);
   
   // Free the allocated memory on both GPU and CPU
   hipFree(d_image);
   delete[] h_image;
}


//*****************************************************************************
// a) cudaEvent
//    Measures the actual execution time of the kernel on the GPU.
//    It excludes data transfer times between the CPU (host) and GPU (device)
//    as well as other initialization operations.
//    It is precise because it directly synchronizes with the GPU.
//
// b) std::chrono
//    Measures the total execution time, including:
//    - Kernel launch (which is typically asynchronous).
//    - Data transfer times between the CPU and GPU.
//    - Any other operations running on the CPU, such as CUDA API calls.
//    - Possible delays caused by synchronization between the CPU and GPU
//    (e.g., hipDeviceSynchronize() waits for all GPU operations to complete).
//*****************************************************************************


int main(int argc, char *argv[])
{
   auto start_chrono = std::chrono::high_resolution_clock::now();
   
   // Check if the correct number of arguments is passed to the program
   if(argc != 4)
   {
      std::cerr << "Usage: " << argv[0] << " <width> <height> <max_iterations> " << std::endl;
      return EXIT_FAILURE;
   }
   
   // Parse command-line arguments for image width, height, and maximum iterations
   int width  = std::stoi(argv[1]); // Image width  in pixels
   int height = std::stoi(argv[2]); // Image height in pixels
   uint32_t max_iterations = std::stoi(argv[3]); // Maximum iterations for Cosine calculation
   
   // Store image size in a structure
   ImageSize size = {width, height};
   
   // Call the function to generate the Cosine fractal on the GPU and save the result as an image
   generateFractal(size, max_iterations);
   
   auto stop_chrono = std::chrono::high_resolution_clock::now();
   
   std::chrono::duration<double, std::milli> elapsed_chrono = stop_chrono - start_chrono;
   std::cout << "Execution time using std::chrono: " << elapsed_chrono.count() << " ms" << std::endl;
   
   return EXIT_SUCCESS;
}
